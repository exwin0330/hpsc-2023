
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__device__ __managed__ int sum, *bucket;

__global__ void zero(int *a) {
  a[threadIdx.x] = 0;
}

__global__ void atomic(int *a, int *key) {
  int n = key[threadIdx.x];
  atomicAdd(&a[n], 1);
}

__global__ void thread(int *key, int n) {
  key[sum + threadIdx.x] = n;
  __syncthreads();
  atomicAdd(&sum, 1);
}

int main() {
  int n = 50;
  int range = 5;
  std::vector<int> key(n);
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  hipMallocManaged(&bucket, range*sizeof(int));
  zero<<<1,range>>>(bucket);
  hipDeviceSynchronize();

  int *keyArray;
  hipMallocManaged(&keyArray, n*sizeof(int));
  std::copy(key.begin(), key.end(), keyArray);
  atomic<<<1,n>>>(bucket, keyArray);
  hipDeviceSynchronize();

  for (int i=0; i<range; i++) {
    thread<<<1,bucket[i]>>>(keyArray, i);
    hipDeviceSynchronize();
  }

  hipFree(bucket);

  for (int i=0; i<n; i++) {
    printf("%d ",keyArray[i]);
  }
  printf("\n");
  hipFree(keyArray);
}
